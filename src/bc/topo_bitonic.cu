#include "hip/hip_runtime.h"
// Copyright (c) 2016, Xuhao Chen
#include "bc.h"
#include "timer.h"
#include "worklistc.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"
#include <vector>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#define BC_VARIANT "topo_bitonic"


// OPT KERNELS BEGIN

__global__ void preprocess1(int N, int *P, bool *isBad, int *CSR_N, int *numBadWarps, int magic_val)
{
    // Set the warp reference array P. count Bad Warps and mark each bad warp in the array isBad.
    // N here is the largest multiple of 32 <= actual N.
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < N)
    {
        int wid = tid / 32;
        if (tid % 32 == 0)
            P[wid] = wid; // set the initial reference
        int max_value = (CSR_N[tid + 1] - CSR_N[tid]);
        int min_value = max_value;
        for (int i = 16; i > 0; i = i / 2)
        {
            max_value = max(max_value, __shfl_down_sync(-1, max_value, i));
            min_value = min(min_value, __shfl_down_sync(-1, min_value, i));
        }
        if (tid % 32 == 0 && max_value - min_value > magic_val) // value based on heuristics!
        {
            isBad[wid] = true;
            atomicAdd(numBadWarps, 1);
        }
    }
}

__global__ void preprocess2(bool *isBad, int badWarps, int *G, int *B, int totalWarps, int *Gi, int *Bi)
{
    // kernel launched with totalWarps number of threads

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    bool flag;
    if (tid < totalWarps)
    {
        flag = isBad[tid];
        if (tid < badWarps)
        {
            if (!flag)
            {
                int i = atomicAdd(Gi, 1);
                G[i] = tid;
            }
        }
        else if (flag)
        {
            int i = atomicAdd(Bi, 1);
            B[i] = tid;
        }
    }
    // launch a dynamic kernel here?
    
}

__global__ void preprocess2_1(int *P, int Gi, int *G, int *B)
{
    // kernel launched with numBadWarps number of threads.

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < Gi)
    {
        int temp1 = G[tid];
        int temp2 = B[tid];
        P[temp1] = temp2;
        P[temp2] = temp1;
    }
}

__global__ void preprocess3(int N, int numBadWarps, int* d_offset, int* thread_map, int* P, int totalWarps){

   __shared__ int ind[512][2];

    unsigned int i, ij, v, wid_orig, wid_new;

    i = threadIdx.x;
    v = i + blockIdx.x * blockDim.x;
    int id = v;

    if (v >= N) return;

    thread_map[v] = v;

    wid_orig = v / 32;
    wid_new = wid_orig;


    if (wid_new < totalWarps){
        wid_new = P[wid_new];
        v = wid_new * 32 + i % 32; // new id according to new warp arrangement
        thread_map[id] = v;
    }

    if ((blockIdx.x * blockDim.x) / 32 < numBadWarps)
    {

        ind[threadIdx.x][0] = (d_offset[v + 1] - d_offset[v]); // the loop's limit
        ind[threadIdx.x][1] = v;

        // if(i == 0) atomicAdd(oblocks,1); // DEBUG STUFF

        for (int k = 2; k <= 512; k <<= 1)
        {
            for (int j = k >> 1; j > 0; j = j >> 1)
            {

                ij = i ^ j;

                if (ij > i)
                {
                    int temp[2];
                    if (((i & k) == 0 && ind[i][0] > ind[ij][0]) || ((i & k) != 0 && ind[i][0] < ind[ij][0]))
                    {
                        temp[0] = ind[i][0];
                        temp[1] = ind[i][1];
                        ind[i][0] = ind[ij][0];
                        ind[i][1] = ind[ij][1];
                        ind[ij][0] = temp[0];
                        ind[ij][1] = temp[1];
                    }
                }
                __syncthreads();
            }
        }

        thread_map[id] = ind[threadIdx.x][1];
    }


}

// OPT KERNELS END



__global__ void initialize(int m, int source, ScoreT *scores, int *path_counts, int *depths, ScoreT *deltas, bool *visited, bool *expanded) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		scores[id] = 0;
		deltas[id] = 0;
		expanded[id] = false;
		if(id == source) {
			visited[id] = true;
			path_counts[id] = 1;
			depths[id] = 0;
		} else {
			visited[id] = false;
			path_counts[id] = 0;
			depths[id] = -1;
		}
	}
}

// Shortest path calculation by forward BFS
__global__ void bc_forward(int m, const int *row_offsets, 
                           const IndexT *column_indices, 
                           int *path_counts, int *depths, int depth, 
                           bool *changed, bool *visited, bool *expanded, 
                           int *nitems, int *queue, int queue_len, int* tmap) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(src < m) src = tmap[src];
	if(src < m && visited[src] && !expanded[src]) {
		expanded[src] = true;
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1]; 
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			if ((depths[dst] == -1) && (atomicCAS(&depths[dst], -1, depth)==-1)) {
				int pos = atomicAdd(nitems, 1);
				queue[queue_len + pos] = dst;
				*changed = true;
			}
			if (depths[dst] == depth) {
				atomicAdd(&path_counts[dst], path_counts[src]);
			}
		}
	}
}

// Dependency accumulation by back propagation
// not sure if optimization should be applied here.
__global__ void bc_reverse(int num, const int *row_offsets, 
                           const IndexT *column_indices, 
                           int start, int *frontiers, 
                           ScoreT *scores, int *path_counts, 
                           int *depths, int depth, ScoreT *deltas) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    
	if(id < num) {
		int src = frontiers[start + id];
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1];
		ScoreT delta_src = 0;
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			if(depths[dst] == depth + 1) {
				delta_src += static_cast<ScoreT>(path_counts[src]) / 
					static_cast<ScoreT>(path_counts[dst]) * (1 + deltas[dst]);
			}
		}
		deltas[src] = delta_src;
		scores[src] += deltas[src];
	}
}

__global__ void bc_update(int m, int *depths, bool *visited) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
        // id = tmap[id];
		if(depths[id] != -1 && !visited[id])
			visited[id] = true;
	}
}

__global__ void bc_normalize(int m, ScoreT *scores, ScoreT max_score) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < m) scores[tid] = scores[tid] / (max_score);
}
void BCSolver(Graph &g, int source, ScoreT *h_scores) {
  auto m = g.V();
  auto nnz = g.E();
  auto h_row_offsets = g.out_rowptr();
  auto h_column_indices = g.out_colidx();	
	//print_device_info(0);
	int zero = 0;
	int *d_row_offsets;
  VertexId *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(VertexId)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(VertexId), hipMemcpyHostToDevice));
	
	ScoreT *d_scores, *d_deltas;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_scores, sizeof(ScoreT) * m));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_deltas, sizeof(ScoreT) * m));
	int *d_path_counts, *d_depths, *d_frontiers;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_path_counts, sizeof(int) * m));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_depths, sizeof(int) * m));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_frontiers, sizeof(int) * (m+1)));
	bool *d_changed, h_changed, *d_visited, *d_expanded;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_visited, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_expanded, m * sizeof(bool)));
	int *d_nitems, h_nitems = 1;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_nitems, sizeof(int)));

	int depth = 0;
	int frontiers_len = 0;
	vector<int> depth_index;
	depth_index.push_back(0);
	int nthreads = 512;
	int nblocks = (m - 1) / nthreads + 1;
	initialize <<<nblocks, nthreads>>> (m, source, d_scores, d_path_counts, d_depths, d_deltas, d_visited, d_expanded);
	// CudaTest("initializing failed");
	CUDA_SAFE_CALL(hipMemcpy(&d_frontiers[0], &source, sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	printf("Launching CUDA BC solver (%d CTAs/SM, %d threads/CTA) ...\n", nblocks, nthreads);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

	
// opt portion

    float milliseconds = 0;

    hipEventRecord(start);

    int *P, *d_P, numBadWarps, *d_numBadWarps;
    bool *isBad, *d_isBad;
    int* thread_mappings;
    int *d_Good, *d_Bad, *d_Gi, *d_Bi;
    int Gi, Bi;
    int totalWarps = (m / 32);
    P = (int *)malloc(totalWarps * sizeof(int));
    isBad = (bool *)malloc(totalWarps);
    hipMalloc(&thread_mappings, m*sizeof(int));
    numBadWarps = 0;
    hipMalloc(&d_numBadWarps, sizeof(int));
    hipMemset(d_numBadWarps, 0, sizeof(int));
    hipMalloc(&d_Gi, sizeof(int));
    hipMalloc(&d_Bi, sizeof(int));
    hipMalloc(&d_P, totalWarps * sizeof(int));

    hipMalloc(&d_isBad, totalWarps);
    hipMemset(d_isBad, 0, totalWarps);

    hipMemset(d_numBadWarps, 0, sizeof(int));
    preprocess1<<<ceil(((float)totalWarps * 32) / 512), 512>>>(totalWarps * 32, d_P, d_isBad, d_row_offsets, d_numBadWarps, 350);
    CUDA_SAFE_CALL(hipMemcpy(&numBadWarps, d_numBadWarps, sizeof(int), hipMemcpyDeviceToHost));
    hipMalloc(&d_Good, numBadWarps * sizeof(int));
    hipMalloc(&d_Bad, numBadWarps * sizeof(int));
    hipMemset(d_Gi, 0, sizeof(int));
    hipMemset(d_Bi, 0, sizeof(int));
    preprocess2<<<ceil(totalWarps * 1.0 / 512), 512>>>(d_isBad, numBadWarps, d_Good, d_Bad, totalWarps, d_Gi, d_Bi);
    CUDA_SAFE_CALL(hipMemcpy(&Gi, d_Gi, sizeof(int), hipMemcpyDeviceToHost));
    preprocess2_1<<<max(1, (int)ceil(Gi * 1.0 / 512)), 512>>>(d_P, Gi, d_Good, d_Bad);


    preprocess3<<<nblocks, nthreads>>>(m, numBadWarps, d_row_offsets, thread_mappings, d_P, totalWarps);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("opt time = %f\n", milliseconds);

    hipEventRecord(start);

	do {
		depth++;
		h_changed = false;
		//printf("iteration=%d, frontire_size=%d\n", depth, h_nitems);
		CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_nitems, &zero, sizeof(int), hipMemcpyHostToDevice));
		frontiers_len += h_nitems;
		depth_index.push_back(frontiers_len);
		bc_forward<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_path_counts, d_depths, depth, d_changed, d_visited, d_expanded, d_nitems, d_frontiers, 
        frontiers_len, thread_mappings);
		// CudaTest("solving bc_forward failed");
		bc_update <<<nblocks, nthreads>>> (m, d_depths, d_visited);
		// CudaTest("solving bc_update failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(&h_nitems, d_nitems, sizeof(int), hipMemcpyDeviceToHost));
	} while (h_changed);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	//printf("\nDone Forward BFS, starting back propagation (dependency accumulation)\n");
	for (int d = depth_index.size() - 2; d >= 0; d--) {
		h_nitems = depth_index[d+1] - depth_index[d];
		//thrust::sort(thrust::device, d_frontiers+depth_index[d], d_frontiers+depth_index[d+1]);
		nblocks = (h_nitems - 1) / nthreads + 1;
		//printf("Reverse: depth=%d, frontier_size=%d\n", d, h_nitems);
		bc_reverse<<<nblocks, nthreads>>>(h_nitems, d_row_offsets, d_column_indices, depth_index[d], d_frontiers, d_scores, d_path_counts, d_depths, d, d_deltas);
		// CudaTest("solving bc_reverse failed");
	}
	
	//CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, sizeof(ScoreT) * m, hipMemcpyDeviceToHost));
	//printf("\nStart calculating the maximum score\n");
	ScoreT *d_max_score;
	d_max_score = thrust::max_element(thrust::device, d_scores, d_scores + m);
	ScoreT h_max_score;
	CUDA_SAFE_CALL(hipMemcpy(&h_max_score, d_max_score, sizeof(ScoreT), hipMemcpyDeviceToHost));
	
	nthreads = 512;
	nblocks = (m - 1) / nthreads + 1;
	bc_normalize<<<nblocks, nthreads>>>(m, d_scores, h_max_score);
	CUDA_SAFE_CALL(hipDeviceSynchronize());


	printf("\titerations = %d.\n", depth);

	hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Runtime = %f\n\n\n", milliseconds);

	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, sizeof(ScoreT) * m, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_path_counts));
	CUDA_SAFE_CALL(hipFree(d_depths));
	CUDA_SAFE_CALL(hipFree(d_deltas));
	CUDA_SAFE_CALL(hipFree(d_frontiers));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
}

