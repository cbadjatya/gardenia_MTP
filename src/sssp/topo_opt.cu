#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#define SSSP_VARIANT "topo_opt"
#include "sssp.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>


// Optimization kernels begin

__global__ void preprocess1(int N, int *P, bool *isBad, uint64_t *CSR_N, int *numBadWarps, int magic_val)
{
    // Set the warp reference array P. count Bad Warps and mark each bad warp in the array isBad.
    // N here is the largest multiple of 32 <= actual N.
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < N)
    {
        int wid = tid / 32;
        if (tid % 32 == 0)
            P[wid] = wid; // set the initial reference
        int max_value = (CSR_N[tid + 1] - CSR_N[tid]);
        int min_value = max_value;
        for (int i = 16; i > 0; i = i / 2)
        {
            max_value = max(max_value, __shfl_down_sync(-1, max_value, i));
            min_value = min(min_value, __shfl_down_sync(-1, min_value, i));
        }
        if (tid % 32 == 0){
            if(max_value - min_value > magic_val){
                isBad[wid] = true;
                atomicAdd(numBadWarps, 1);
            }
            else{
                isBad[wid] = false;
            }
        }
    }
}

__global__ void preprocess2(bool *isBad, int badWarps, int *G, int *B, int totalWarps, int *Gi, int *Bi)
{
    // kernel launched with totalWarps number of threads

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    bool flag;
    if (tid < totalWarps)
    {
        flag = isBad[tid];
        if (tid < badWarps)
        {
            if (!flag)
            {
                int i = atomicAdd(Gi, 1);
                G[i] = tid;
            }
        }
        else if (flag)
        {
            int i = atomicAdd(Bi, 1);
            B[i] = tid;
        }
    }
    // launch a dynamic kernel here?
    
}

__global__ void preprocess2_1(int *P, int Gi, int *G, int *B)
{
    // kernel launched with numBadWarps number of threads.

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < Gi)
    {
        int temp1 = G[tid];
        int temp2 = B[tid];
        P[temp1] = temp2;
        P[temp2] = temp1;
    }
}

__global__ void preprocess3(int N, int numBadWarps, uint64_t* d_offset, int* thread_map, int* P, int totalWarps){

   __shared__ unsigned ind[512][2];

    unsigned int i, ij, v, wid_new;

    i = threadIdx.x;
    v = i + blockIdx.x * blockDim.x;
    unsigned int id = v;

    if (v >= N) return;

    thread_map[v] = v;
    wid_new = v / 32;

    if (wid_new < totalWarps){
        wid_new = P[wid_new];
        v = wid_new * 32 + i % 32; // new id according to new warp arrangement
        thread_map[id] = v;
    }

    return;


    if ((blockIdx.x * blockDim.x) / 32 < numBadWarps)
    {

        ind[threadIdx.x][0] = (d_offset[v + 1] - d_offset[v]); // the loop's limit
        ind[threadIdx.x][1] = v;

        // if(i == 0) atomicAdd(oblocks,1); // DEBUG STUFF

        for (int k = 2; k <= 512; k <<= 1)
        {
            for (int j = k >> 1; j > 0; j = j >> 1)
            {

                ij = i ^ j;

                if (ij > i)
                {
                    int temp[2];
                    if (((i & k) == 0 && ind[i][0] > ind[ij][0]) || ((i & k) != 0 && ind[i][0] < ind[ij][0]))
                    {
                        temp[0] = ind[i][0];
                        temp[1] = ind[i][1];
                        ind[i][0] = ind[ij][0];
                        ind[i][1] = ind[ij][1];
                        ind[ij][0] = temp[0];
                        ind[ij][1] = temp[1];
                    }
                }
                __syncthreads();
            }
        }

        thread_map[id] = ind[threadIdx.x][1];
    }
}

// keeping grp_cnt global right now, planning to do a complete sort (for bad blocks)

// for each warp, split into groups based on workload of the constituting threads.
// The sort will then be applied on all groups. 
// do I need to use newId for indexing anything except offsets?

// do global variables need to be made volatile? 

__global__ void preprocess4(int* thread_map, uint64_t* d_offsets, unsigned int* grp_cnt, int* threadToGroup, int* localIndexInGroup, int* group_map, int* group_size, int* group_load, int numBadWarps
                                , int GROUP_THRESHOLD){

    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id >= numBadWarps* 32) return;

    int node_to_be_processed = thread_map[id];


    if(threadIdx.x % 32 == 0){
        
        int curr_sz = 1;
        int curr_load = d_offsets[node_to_be_processed+1] - d_offsets[node_to_be_processed];
        int curr_groupId = atomicInc(grp_cnt, INT_MAX);
        threadToGroup[id] = curr_groupId;
        localIndexInGroup[id] = 0;

        for(int i = 1; i < 32; i++){

            int load_i = d_offsets[node_to_be_processed + 1 + i] - d_offsets[node_to_be_processed + i];

            if(curr_load > 32 || curr_load + load_i > GROUP_THRESHOLD){
                group_map[curr_groupId] = curr_groupId; // do we need this??
                group_size[curr_groupId] = curr_sz;
                group_load[curr_groupId] = curr_load;

                curr_load = 0;
                curr_groupId = atomicInc(grp_cnt, INT_MAX);
                curr_sz = 0;
            }

            curr_load += load_i;
            curr_sz++;
            threadToGroup[id + i] = curr_groupId; // using new_Id here. Correct since local mapping for threads within a warp is still the same.
            localIndexInGroup[id + i] = curr_sz - 1;

        }

        group_map[curr_groupId] = curr_groupId;
        group_size[curr_groupId] = curr_sz;
        group_load[curr_groupId] = curr_load;
    }
    
}

// do I need to sort groupMap or simply sort threadToGroupMapping based on the size...but would it be a key value pair mapping??

// calculate a prefix sum with the sizes of each group pref[grp]

// once all the groups have been sorted, id needs to be reassigned from thread_map[id] to 
// pref[group_map[threadToGroup[thread_map[id]]]] - group_size[group_map[threadToGroup[thread_map[id]]]] + localIndexInGroup[new_id]
// do I actually need to sort group_size too??

__global__ void preprocess5(int* thread_map, int numBadWarps, int* threadToGroup, int* r_map, int* pref, int* group_size, int* localIndexInGroup){
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    if( id >= numBadWarps* 32) return;

    int old_grp = threadToGroup[id];
    int new_group = r_map[old_grp];
    int grpPos = pref[new_group] - group_size[new_group]; // sizes need to be sorted to calculate prefix sum...
    
    thread_map[grpPos + localIndexInGroup[id]] = thread_map[id];

}

__global__ void make_rev_map(int* rmap, int* gmap, int N){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < N){
        rmap[gmap[id]] = id;
    }
}

// OPT KERNELS END

//Naive CUDA implementation of the Bellman-Ford algorithm for SSSP
__global__ void initialize(int m, int source, bool *visited, bool *expanded) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		expanded[id] = false;
		if(id == source) visited[id] = true;
		else visited[id] = false;
	}
}

/**
 * @brief naive Bellman_Ford SSSP kernel entry point.
 *
 * @param[in] m                 Number of vertices
 * @param[in] d_row_offsets     Device pointer of VertexId to the row offsets queue
 * @param[in] d_column_indices  Device pointer of VertexId to the column indices queue
 * @param[in] d_weight          Device pointer of DistT to the edge weight queue
 * @param[out]d_dist            Device pointer of DistT to the distance queue
 * @param[in] d_in_queue        Device pointer of VertexId to the incoming frontier queue
 * @param[out]d_out_queue       Device pointer of VertexId to the outgoing frontier queue
 */
__global__ void bellman_ford(int m, uint64_t *row_offsets, VertexId *column_indices, DistT *weight, DistT *dist, bool *changed, bool *visited, bool *expanded, int* tmap) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
    int id = src;
	if(src >= m) return;
    
    // preventing unnecessary access?? Doesn't seem like it...
    // if(blockIdx.x < numBadBlocks || isBad[src/32])
    src = tmap[src];

	if(visited[src] && !expanded[src]) { // visited but not expanded
		expanded[src] = true;
		//atomicAdd(num_frontier, 1);
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src + 1];
        int dsrc = dist[src];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			DistT old_dist = dist[dst]; 
			DistT new_dist = dsrc + weight[offset];
			if (new_dist < old_dist) {
				if (atomicMin(&dist[dst], new_dist) > new_dist) {
					if(expanded[dst]) expanded[dst] = false;
					*changed = true;
				}
			}
		}
	}
}

__global__ void update(int m, DistT *dist, bool *visited) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		if(dist[id] < MYINFINITY && !visited[id])
			visited[id] = true;
	}
}

/**
 * @brief naive topology-driven mapping GPU SSSP entry point.
 *
 * @param[in] m                 Number of vertices
 * @param[in] h_row_offsets     Host pointer of VertexId to the row offsets queue
 * @param[in] h_column_indices  Host pointer of VertexId to the column indices queue
 * @param[in] h_weight          Host pointer of DistT to the edge weight queue
 * @param[out]h_dist            Host pointer of DistT to the distance queue
 */
void SSSPSolver(Graph &g, int source, DistT *h_weight, DistT *h_dist, int delta, int magic) {
	auto m = g.V();
	auto nnz = g.E();
	auto h_row_offsets = g.out_rowptr();
	auto h_column_indices = g.out_colidx();	
	//print_device_info(0);
	uint64_t *d_row_offsets;
	VertexId *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(uint64_t)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(VertexId)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(uint64_t), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(VertexId), hipMemcpyHostToDevice));

	DistT zero = 0;
	int one = 1;
	DistT *d_weight;
	DistT * d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_weight, nnz * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_weight, h_weight, nnz * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(&d_dist[source], &zero, sizeof(zero), hipMemcpyHostToDevice));
	
	bool *d_changed, h_changed, *d_visited, *d_expanded;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_visited, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_expanded, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemset(d_visited, 0, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemcpy(&d_visited[source], &one, sizeof(bool), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemset(d_expanded, 0, m * sizeof(bool)));

	CUDA_SAFE_CALL(hipDeviceSynchronize());

	// printf("Source node neighbors : %d\n", h_row_offsets[source+1] - h_row_offsets[source]);

	
	int iter = 0;
	//int h_num_frontier = 1;
	int nthreads = 512;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA SSSP solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);


	// Timer t1;
	// t1.Start();
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start);
    // Adding OPT Code

    int *P, *d_P, numBadWarps, *d_numBadWarps;
    bool *isBad, *d_isBad;
    int* thread_mappings;
    int *d_Good, *d_Bad, *d_Gi, *d_Bi;
    int Gi, Bi;
    int totalWarps = (m / 32);
    P = (int *)malloc(totalWarps * sizeof(int));
    isBad = (bool *)malloc(totalWarps);
    hipMalloc(&thread_mappings, m*sizeof(int));
    numBadWarps = 0;
    hipMalloc(&d_numBadWarps, sizeof(int));
    hipMemset(d_numBadWarps, 0, sizeof(int));
    hipMalloc(&d_Gi, sizeof(int));
    hipMalloc(&d_Bi, sizeof(int));
    hipMalloc(&d_P, totalWarps * sizeof(int));

    hipMalloc(&d_isBad, totalWarps);
    hipMemset(d_isBad, 0, totalWarps);

    hipMemset(d_numBadWarps, 0, sizeof(int));
    preprocess1<<<ceil(((float)totalWarps * 32) / 512), 512>>>(totalWarps * 32, d_P, d_isBad, d_row_offsets, d_numBadWarps, magic);
    CUDA_SAFE_CALL(hipMemcpy(&numBadWarps, d_numBadWarps, sizeof(int), hipMemcpyDeviceToHost));
    hipMalloc(&d_Good, numBadWarps * sizeof(int));
    hipMalloc(&d_Bad, numBadWarps * sizeof(int));
    hipMemset(d_Gi, 0, sizeof(int));
    hipMemset(d_Bi, 0, sizeof(int));
    preprocess2<<<ceil(totalWarps * 1.0 / 512), 512>>>(d_isBad, numBadWarps, d_Good, d_Bad, totalWarps, d_Gi, d_Bi);
    CUDA_SAFE_CALL(hipMemcpy(&Gi, d_Gi, sizeof(int), hipMemcpyDeviceToHost));
    preprocess2_1<<<max(1, (int)ceil(Gi * 1.0 / 512)), 512>>>(d_P, Gi, d_Good, d_Bad);


    preprocess3<<<nblocks, nthreads>>>(m, numBadWarps, d_row_offsets, thread_mappings, d_P, totalWarps);

     int* hmap;
    int rmpd = numBadWarps*32;

    hmap = (int*)malloc(sizeof(int) * (rmpd + 1));
    hipMemcpy(hmap, thread_mappings, (rmpd+1)* sizeof(int), hipMemcpyDeviceToHost);

    unsigned int* grp_cnt;
    int* threadToGroup;
    int* localIndexInGroup;
    int* group_map;
    int* group_size;
    int* group_load;
    int* pref;
    int* h_group_load, *h_group_map, *h_group_size, *h_ttg, * h_lind;


    hipMalloc(&grp_cnt, sizeof(int));
    hipMalloc(&threadToGroup, numBadWarps*32*sizeof(int));
    hipMalloc(&localIndexInGroup, numBadWarps*32*sizeof(int));
    hipMalloc(&group_map, numBadWarps*32*sizeof(int));
    hipMalloc(&group_size, numBadWarps*32*sizeof(int));
    hipMalloc(&group_load, numBadWarps*32*sizeof(int));

    


    hipMemset(grp_cnt, 0, sizeof(int));

    

    int p4block_size = 512;
    int p4blocks = (numBadWarps*32 - 1)/512 + 1;

    preprocess4<<<p4blocks, p4block_size>>>(thread_mappings, d_row_offsets, grp_cnt, threadToGroup, localIndexInGroup, group_map, group_size, group_load, numBadWarps, 50);



    int h_grp_cnt;
    hipMemcpy(&h_grp_cnt, grp_cnt, sizeof(int), hipMemcpyDeviceToHost);


    auto zipIt = thrust::make_zip_iterator(thrust::make_tuple(group_map, group_size));

    thrust::sort_by_key(thrust::device, group_load, group_load+h_grp_cnt, zipIt);

    // calculating prefix sum could be faster on CPU??
    hipMalloc(&pref, h_grp_cnt*sizeof(int));
    hipMemcpy(pref, group_size, h_grp_cnt * sizeof(int), hipMemcpyDeviceToDevice);
    thrust::inclusive_scan(thrust::device, pref, pref+h_grp_cnt, pref);

    // make reverse group mapping
    int* r_map;
    hipMalloc(&r_map, sizeof(int)* h_grp_cnt);
    make_rev_map<<<(h_grp_cnt - 1)/512 + 1, 512>>>(r_map, group_map, h_grp_cnt);

    preprocess5<<<p4blocks, p4block_size>>>(thread_mappings, numBadWarps, threadToGroup, r_map, pref, group_size, localIndexInGroup);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);

    printf("time taken for opt = %f\n",elapsed);
    printf("Number of Bad Warps found = %d\n", numBadWarps);

	// Timer t;
	// t.Start();

    hipEventRecord(start, 0);


	do {
		++ iter;
		h_changed = false;
		CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));
		//CUDA_SAFE_CALL(hipMemcpy(d_num_frontier, &zero, sizeof(int), hipMemcpyHostToDevice));
		// hipProfilerStart();
		bellman_ford<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_weight, d_dist, d_changed, d_visited, d_expanded, thread_mappings);
		// hipProfilerStop();
		update<<<nblocks, nthreads>>>(m, d_dist, d_visited);
		// CudaTest("solving failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(&h_num_frontier, d_num_frontier, sizeof(int), hipMemcpyDeviceToHost));
		//printf("iteration %d: num_frontier = %d\n", iter, h_num_frontier);
	} while (h_changed);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	// t.Stop();

	hipEventRecord(stop);
    hipEventSynchronize(stop);
    elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", SSSP_VARIANT, elapsed);

	CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_weight));
	CUDA_SAFE_CALL(hipFree(d_dist));
	CUDA_SAFE_CALL(hipFree(d_changed));
	// CUDA_SAFE_CALL(hipFree(d_num_frontier));
	return;
}
