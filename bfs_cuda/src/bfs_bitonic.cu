#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

// bitonic code

#define NUM_TESTS 1

#define CHECK(call) \
    {               \
        call;       \
    }

#define CHECK_DEBUG(call)                                                         \
    {                                                                             \
        const hipError_t error = call;                                           \
        if (error != hipSuccess)                                                 \
        {                                                                         \
            printf("Error : %s: %d -> ", __FILE__, __LINE__);                     \
            printf("code : %d, reason : %s\n", error, hipGetErrorString(error)); \
        }                                                                         \
    }

using namespace std;

__global__ void preprocess1(int N, int *P, bool *isBad, int *CSR_N, int *numBadWarps, int magic_val)
{
    // Set the warp reference array P. count Bad Warps and mark each bad warp in the array isBad.
    // N here is the largest multiple of 32 <= actual N.
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < N)
    {
        int wid = tid / 32;
        if (tid % 32 == 0)
            P[wid] = wid; // set the initial reference
        int max_value = (CSR_N[tid + 1] - CSR_N[tid]);
        int min_value = max_value;
        for (int i = 16; i > 0; i = i / 2)
        {
            max_value = max(max_value, __shfl_down_sync(-1, max_value, i));
            min_value = min(min_value, __shfl_down_sync(-1, min_value, i));
        }
        if (tid % 32 == 0){
            if(max_value - min_value > magic_val){
                isBad[wid] = true;
                atomicAdd(numBadWarps, 1);
            }
            else{
                isBad[wid] = false;
            }
        }
    }
}

__global__ void preprocess2(bool *isBad, int badWarps, int *G, int *B, int totalWarps, int *Gi, int *Bi)
{
    // kernel launched with totalWarps number of threads

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    bool flag;
    if (tid < totalWarps)
    {
        flag = isBad[tid];
        if (tid < badWarps)
        {
            if (!flag)
            {
                int i = atomicAdd(Gi, 1);
                G[i] = tid;
            }
        }
        else if (flag)
        {
            int i = atomicAdd(Bi, 1);
            B[i] = tid;
        }
    }
    // launch a dynamic kernel here?
    
}

__global__ void preprocess2_1(int *P, int Gi, int *G, int *B)
{
    // kernel launched with numBadWarps number of threads.

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < Gi)
    {
        int temp1 = G[tid];
        int temp2 = B[tid];
        P[temp1] = temp2;
        P[temp2] = temp1;
    }
}

__global__ void preprocess3(int N, int numBadWarps, int* d_offset, int* thread_map, int* P, int totalWarps){

   __shared__ unsigned ind[512][2];

    unsigned int i, ij, v, wid_new;

    i = threadIdx.x;
    v = i + blockIdx.x * blockDim.x;
    unsigned int id = v;

    if (v >= N) return;

    thread_map[v] = v;
    wid_new = v / 32;

    if (wid_new < totalWarps){
        wid_new = P[wid_new];
        v = wid_new * 32 + i % 32; // new id according to new warp arrangement
        thread_map[id] = v;
    }

    if ((blockIdx.x * blockDim.x) / 32 < numBadWarps)
    {

        ind[threadIdx.x][0] = (d_offset[v + 1] - d_offset[v]); // the loop's limit
        ind[threadIdx.x][1] = v;

        // if(i == 0) atomicAdd(oblocks,1); // DEBUG STUFF

        for (int k = 2; k <= 512; k <<= 1)
        {
            for (int j = k >> 1; j > 0; j = j >> 1)
            {

                ij = i ^ j;

                if (ij > i)
                {
                    int temp[2];
                    if (((i & k) == 0 && ind[i][0] > ind[ij][0]) || ((i & k) != 0 && ind[i][0] < ind[ij][0]))
                    {
                        temp[0] = ind[i][0];
                        temp[1] = ind[i][1];
                        ind[i][0] = ind[ij][0];
                        ind[i][1] = ind[ij][1];
                        ind[ij][0] = temp[0];
                        ind[ij][1] = temp[1];
                    }
                }
                __syncthreads();
            }
        }

        thread_map[id] = ind[threadIdx.x][1];
    }
}

// OPT KERNELS END


__global__ void initialize_graph(int N, int *parent, int *phase, int ROOT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        parent[i] = -1;
        phase[i] = -1;
        if (i == ROOT)
        {
            phase[ROOT] = 0;
        }
    }
}


__global__ void broadcast_graph(int p, int N, int *phase, int *CSR_N, int *CSR_F, int *parent, int *discovered, int* thread_mappings)
{

    unsigned id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id >= N) return;

    id = thread_mappings[id];

    if (phase[id] == p)
        for (int j =  CSR_N[id]; j < CSR_N[id + 1]; j++)
        {
            int nbr = CSR_F[j];
            if (phase[nbr] < 0)
            {
                phase[nbr] = p + 1;
                parent[nbr] = id;
                *discovered = 0;
            }
        }
}

void printSolution(int *phase, int *parent, int N)
{
    for (int i = 0; i < N; i++)
    {
        printf("node %d -> parent = %d, phase = %d\n", i, parent[i], phase[i]);
    }
}   

void readGraph(char *filename, int *N, int *ROOT, int **CSR_N, int **CSR_F, int *N_sz, int *F_sz)
{ // verified

    FILE *in = fopen(filename, "r");
    fscanf(in, "%d\n", N);
    fscanf(in, "%d\n", N_sz);
    fscanf(in, "%d\n", F_sz);
    fscanf(in, "%d\n", ROOT);
    *CSR_N = (int *)malloc(*N_sz * sizeof(int));
    *CSR_F = (int *)malloc(*F_sz * sizeof(int));

    for (int i = 0; i < *N_sz; i++)
        fscanf(in, "%d ", &((*CSR_N)[i]));
    for (int i = 0; i < *F_sz; i++)
        fscanf(in, "%d ", &((*CSR_F)[i]));

    fclose(in);
}

int main(int argc, char *argv[])
{
    int N, ROOT;
    int N_sz, F_sz;
    int *CSR_F;
    int *phase, *CSR_N, p;

    readGraph(argv[1], &N, &ROOT, &CSR_N, &CSR_F, &N_sz, &F_sz);
    int *parent, *parent_d, *phase_d;
    int *CSR_N_d, *CSR_F_d;
    parent = (int *)malloc(N * sizeof(int));
    phase = (int *)malloc(N * sizeof(int));

    for (int i = 0; i < NUM_TESTS; i++)
    {

        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipMalloc((void **)&parent_d, N * sizeof(int));
        hipMalloc((void **)&phase_d, N * sizeof(int));
        hipMalloc((void **)&CSR_N_d, N_sz * sizeof(int));
        hipMalloc((void **)&CSR_F_d, F_sz * sizeof(int));

        hipMemcpy(CSR_N_d, CSR_N, N_sz * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(CSR_F_d, CSR_F, F_sz * sizeof(int), hipMemcpyHostToDevice);

        p = 0;

        int discovered = 1;
        int *discovered_d;

        hipMalloc((void **)&discovered_d, sizeof(int));
        hipMemcpy(discovered_d, &discovered, sizeof(int), hipMemcpyHostToDevice);

        int num_threads = 512;
        int num_blocks = ceil((N * 1.0) / num_threads);

        initialize_graph<<<num_blocks, num_threads>>>(N, parent_d, phase_d, ROOT);
        hipError_t err = hipGetLastError(); // Get error code

        if (err != hipSuccess)
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();

        discovered = 0;
        float milliseconds = 0;


        hipEventRecord(start);
        int m = N;
        int* d_row_offsets = CSR_N_d;

        int *P, *d_P, numBadWarps, *d_numBadWarps;
        bool *isBad, *d_isBad;
        int* thread_mappings;
        int *d_Good, *d_Bad, *d_Gi, *d_Bi;
        int Gi, Bi;
        int totalWarps = (m / 32);
        P = (int *)malloc(totalWarps * sizeof(int));
        isBad = (bool *)malloc(totalWarps);
        hipMalloc(&thread_mappings, m*sizeof(int));
        numBadWarps = 0;
        hipMalloc(&d_numBadWarps, sizeof(int));
        hipMemset(d_numBadWarps, 0, sizeof(int));
        hipMalloc(&d_Gi, sizeof(int));
        hipMalloc(&d_Bi, sizeof(int));
        hipMalloc(&d_P, totalWarps * sizeof(int));

        hipMalloc(&d_isBad, totalWarps);
        hipMemset(d_isBad, 0, totalWarps);

        hipMemset(d_numBadWarps, 0, sizeof(int));
        preprocess1<<<ceil(((float)totalWarps * 32) / 512), 512>>>(totalWarps * 32, d_P, d_isBad, d_row_offsets, d_numBadWarps, 350);
        CHECK(hipMemcpy(&numBadWarps, d_numBadWarps, sizeof(int), hipMemcpyDeviceToHost));
        hipMalloc(&d_Good, numBadWarps * sizeof(int));
        hipMalloc(&d_Bad, numBadWarps * sizeof(int));
        hipMemset(d_Gi, 0, sizeof(int));
        hipMemset(d_Bi, 0, sizeof(int));
        preprocess2<<<ceil(totalWarps * 1.0 / 512), 512>>>(d_isBad, numBadWarps, d_Good, d_Bad, totalWarps, d_Gi, d_Bi);
        CHECK(hipMemcpy(&Gi, d_Gi, sizeof(int), hipMemcpyDeviceToHost));
        preprocess2_1<<<max(1, (int)ceil(Gi * 1.0 / 512)), 512>>>(d_P, Gi, d_Good, d_Bad);


        preprocess3<<<num_blocks, num_threads>>>(m, numBadWarps, d_row_offsets, thread_mappings, d_P, totalWarps);


       
        

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
         
        printf("opt time = %f\n", milliseconds);
        milliseconds = 0;
        hipEventRecord(start);


        while (!discovered)
        {

            discovered = 1;
            hipMemcpy(discovered_d, &discovered, sizeof(int), hipMemcpyHostToDevice);

            broadcast_graph<<<num_blocks, num_threads>>>(p, N, phase_d, CSR_N_d, CSR_F_d, parent_d, discovered_d, thread_mappings);

            CHECK(hipMemcpy(&discovered, discovered_d, sizeof(int), hipMemcpyDeviceToHost));
            p++;
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Runtime = %f\n", milliseconds);

        printf("------------------------Total no. of phases : %d ------------------------\n\n\n", p);
        // printf("Avg time taken by the kernels : %f ms\n\n", avg_time / iter);
        hipMemcpy(phase, phase_d, N * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(parent, parent_d, N * sizeof(int), hipMemcpyDeviceToHost);
        // fprintf(stderr,"%d %d %d\n",phase, parent[ROOT], phase[ROOT]);
       // printSolution(phase, parent, N);
    }
    return 0;
}

