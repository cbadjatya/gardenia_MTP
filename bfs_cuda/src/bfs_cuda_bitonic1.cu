#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#define NUM_TESTS 1

using namespace std;


// using global definitions for better sorting
int* CSR_N, *phase, p;

__global__ void initialize_graph(int N, int* parent, int* phase, int ROOT){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<N){
        parent[i] = -1;
        phase[i] = -1;
        if(i==ROOT){
            phase[ROOT] = 0;
        }
    }
}


__global__ void broadcast_graph(int p, int N, int* phase, int* CSR_N, int* CSR_F, int* parent, int* discovered){


    // instead of just one warp playing, how about there are at least 4 to tango?                           
     // Remapping / Balancing Code ---- apply only if this is a full block. If N - blockIdx.x * blockDim.x < 512, go forth with a normal implemementation.
     // let the flag be int and be updated by atomics, turn the opt on if it is at least 4 (or some other heuristic)
    unsigned int i, ij, id, size;
	__shared__ int ind[512][2];
	__shared__ int flag; 

    i = threadIdx.x;
	id = i + blockIdx.x * blockDim.x;
	
	if(i == 0) flag = 0;
	size = 512;
	if(N - blockIdx.x * blockDim.x >= 512){
		
   		 
		ind[threadIdx.x][0] = (phase[id]==p)*(CSR_N[id+1]-CSR_N[id]); //the loop's limit
    	ind[threadIdx.x][1] = id;
		int max_value = ind[i][0];
		int min_value = ind[i][0];
		for (int i=16; i>0; i=i/2){
			max_value = max(max_value, __shfl_down_sync(-1, max_value, i));
			min_value = min(min_value, __shfl_down_sync(-1, min_value, i));
		}
		int warp_score = max_value - min_value;
		if(warp_score > 700 && threadIdx.x % 32 == 0) atomicAdd(&flag,1); // if any one of the warp threads meet the condition do the deed.
	}
   
   __syncthreads();
   if(flag >= 2)
    for(int k = 2; k<= size; k<<=1){
      for(int j = k>>1; j > 0; j = j >> 1){

           ij = i ^ j;

           if (ij > i)
           {   
               int temp[2];
               if (((i & k) == 0 && ind[i][0] > ind[ij][0]) ||((i & k) != 0 && ind[i][0] < ind[ij][0]))
               {
                       temp[0] = ind[i][0]; temp[1] = ind[i][1];
                       ind[i][0] = ind[ij][0]; ind[i][1] = ind[ij][1];
                       ind[ij][0] = temp[0]; ind[ij][1] = temp[1];
               }
           }
           __syncthreads();
       }
    }
   
    if(N - blockIdx.x * blockDim.x >= 512){
		id = ind[threadIdx.x][1];
	}
	
	if(id < N){
		int lim = (phase[id]==p)*(CSR_N[id+1]-CSR_N[id]);
		for(int j = 0; j < lim; j++){
			int nbr = CSR_F[j+CSR_N[id]];
			if(phase[nbr] < 0){
				phase[nbr] = p+1;
				parent[nbr] = id;
				*discovered = 0;
			
			}
		}
//		for(int j = 0; j < (phase[id]==p)*(CSR_N[id+1]-CSR_N[id])*100; j++){
//			int x = 20;
//			x = x + 10;
//			// useless operations.
//		}
		
	}

}


void printSolution(int* phase ,int* parent, int N){
	for(int i=0;i<N;i++){
		printf("node %d -> parent = %d, phase = %d\n",i,parent[i],phase[i]);
	}
}

void readGraph(char* filename, int* N, int* ROOT, int** CSR_N, int** CSR_F, int* N_sz, int* F_sz){ //verified

	  FILE* in = fopen(filename, "r");
    fscanf(in, "%d\n", N);
    fscanf(in, "%d\n", N_sz);
    fscanf(in, "%d\n", F_sz);
    fscanf(in, "%d\n", ROOT);
    *CSR_N = (int*) malloc(*N_sz*sizeof(int));
    *CSR_F = (int*) malloc(*F_sz*sizeof(int));

    for(int i=0;i<*N_sz;i++) fscanf(in, "%d ", &((*CSR_N)[i]));
    for(int i=0;i<*F_sz;i++) fscanf(in, "%d ", &((*CSR_F)[i]));

    fclose(in);

}

int main(int argc, char* argv[]){
	int N,ROOT;
	int N_sz, F_sz;
        int* CSR_F;
    int* CSR_N, *phase, p;

	readGraph(argv[1], &N, &ROOT, &CSR_N, &CSR_F, &N_sz, &F_sz);
	int* parent, *parent_d, *phase_d;
	int* CSR_N_d, *CSR_F_d;
	parent = (int*)malloc(N*sizeof(int));
	phase = (int*)malloc(N*sizeof(int));

    for(int i=0;i<NUM_TESTS;i++)
    {
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipMalloc((void**)&parent_d, N*sizeof(int));
        hipMalloc((void**)&phase_d, N*sizeof(int));
        hipMalloc((void**)&CSR_N_d, N_sz*sizeof(int));
        hipMalloc((void**)&CSR_F_d, F_sz*sizeof(int));


        hipMemcpy(CSR_N_d, CSR_N, N_sz*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(CSR_F_d, CSR_F, F_sz*sizeof(int), hipMemcpyHostToDevice);

        p = 0;

        int discovered = 1;
        int* discovered_d;

        hipMalloc((void**)&discovered_d, sizeof(int));
        hipMemcpy(discovered_d, &discovered, sizeof(int), hipMemcpyHostToDevice);

        int num_threads = 512;
        int num_blocks = ceil((N*1.0)/num_threads);

        initialize_graph<<< num_blocks, num_threads >>> (N, parent_d, phase_d, ROOT);
        hipError_t err = hipGetLastError();        // Get error code

        if ( err != hipSuccess )
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();


        discovered = 0;
        int iter = 0;
        float avg_time = 0;
        float milliseconds = 0;

        hipEventRecord(start);
        while(!discovered){

            discovered = 1;
            hipMemcpy(discovered_d, &discovered, sizeof(int), hipMemcpyHostToDevice);

            broadcast_graph <<<num_blocks, num_threads>>> (p, N, phase_d, CSR_N_d, CSR_F_d, parent_d, discovered_d);
            //cudaDeviceSynchronize();
           // if ( err != cudaSuccess )
           // {
            //    printf("CUDA Error: %s\n", cudaGetErrorString(err));
             //   exit(-1);
           // }		
            hipMemcpy(&discovered, discovered_d, sizeof(int), hipMemcpyDeviceToHost);
            
            p++;


            hipMemcpy(phase, phase_d, N*sizeof(int), hipMemcpyDeviceToHost);

            //printf("%d ",p);
        }
        //printf("\n");
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        //t = clock() - t;
        //double time_taken = ((double)t)/CLOCKS_PER_SEC;
        printf("%f\n",milliseconds);

        //printf("Total no. of phases : %d\n",iter);  
        //printf("Avg time taken by the kernels : %f ms\n\n",avg_time/iter);
        hipMemcpy(phase, phase_d, N*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(parent, parent_d, N*sizeof(int), hipMemcpyDeviceToHost);
        fprintf(stderr,"%d %d %d\n",phase, parent[ROOT], phase[ROOT]);
        //printSolution(phase, parent, N);
    }
	return 0;
}
