#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>

#define NUM_TESTS 1

#define CHECK(call) \
    {               \
        call;       \
    }

#define CHECK_DEBUG(call)                                                         \
    {                                                                             \
        const hipError_t error = call;                                           \
        if (error != hipSuccess)                                                 \
        {                                                                         \
            printf("Error : %s: %d -> ", __FILE__, __LINE__);                     \
            printf("code : %d, reason : %s\n", error, hipGetErrorString(error)); \
        }                                                                         \
    }

using namespace std;

__global__ void preprocess1(int N, int *P, bool *isBad, int *CSR_N, int *numBadWarps, int magic_val)
{
    // Set the warp reference array P. count Bad Warps and mark each bad warp in the array isBad.
    // N here is the largest multiple of 32 <= actual N.
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < N)
    {
        int wid = tid / 32;
        if (tid % 32 == 0)
            P[wid] = wid; // set the initial reference
        int max_value = (CSR_N[tid + 1] - CSR_N[tid]);
        int min_value = max_value;
        for (int i = 16; i > 0; i = i / 2)
        {
            max_value = max(max_value, __shfl_down_sync(-1, max_value, i));
            min_value = min(min_value, __shfl_down_sync(-1, min_value, i));
        }
        if (tid % 32 == 0){
            if(max_value - min_value > magic_val){
                isBad[wid] = true;
                atomicAdd(numBadWarps, 1);
            }
            else{
                isBad[wid] = false;
            }
        }
    }
}

__global__ void preprocess2(bool *isBad, int badWarps, int *G, int *B, int totalWarps, int *Gi, int *Bi)
{
    // kernel launched with totalWarps number of threads

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    bool flag;
    if (tid < totalWarps)
    {
        flag = isBad[tid];
        if (tid < badWarps)
        {
            if (!flag)
            {
                int i = atomicAdd(Gi, 1);
                G[i] = tid;
            }
        }
        else if (flag)
        {
            int i = atomicAdd(Bi, 1);
            B[i] = tid;
        }
    }
    // launch a dynamic kernel here?
    
}

__global__ void preprocess2_1(int *P, int Gi, int *G, int *B)
{
    // kernel launched with numBadWarps number of threads.

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < Gi)
    {
        int temp1 = G[tid];
        int temp2 = B[tid];
        P[temp1] = temp2;
        P[temp2] = temp1;
    }
}

__global__ void preprocess3(int N, int numBadWarps, int* d_offset, int* thread_map, int* P, int totalWarps){

   __shared__ unsigned ind[512][2];

    unsigned int i, ij, v, wid_new;

    i = threadIdx.x;
    v = i + blockIdx.x * blockDim.x;
    unsigned int id = v;

    if (v >= N) return;

    thread_map[v] = v;
    wid_new = v / 32;

    if (wid_new < totalWarps){
        wid_new = P[wid_new];
        v = wid_new * 32 + i % 32; // new id according to new warp arrangement
        thread_map[id] = v;
    }


    return;



    if ((blockIdx.x * blockDim.x) / 32 < numBadWarps)
    {

        ind[threadIdx.x][0] = (d_offset[v + 1] - d_offset[v]); // the loop's limit
        ind[threadIdx.x][1] = v;

        // if(i == 0) atomicAdd(oblocks,1); // DEBUG STUFF

        for (int k = 2; k <= 512; k <<= 1)
        {
            for (int j = k >> 1; j > 0; j = j >> 1)
            {

                ij = i ^ j;

                if (ij > i)
                {
                    int temp[2];
                    if (((i & k) == 0 && ind[i][0] > ind[ij][0]) || ((i & k) != 0 && ind[i][0] < ind[ij][0]))
                    {
                        temp[0] = ind[i][0];
                        temp[1] = ind[i][1];
                        ind[i][0] = ind[ij][0];
                        ind[i][1] = ind[ij][1];
                        ind[ij][0] = temp[0];
                        ind[ij][1] = temp[1];
                    }
                }
                __syncthreads();
            }
        }

        thread_map[id] = ind[threadIdx.x][1];
    }
}

// keeping grp_cnt global right now, planning to do a complete sort (for bad blocks)

// for each warp, split into groups based on workload of the constituting threads.
// The sort will then be applied on all groups. 
// do I need to use newId for indexing anything except offsets?

// do global variables need to be made volatile? 

__global__ void preprocess4(int* thread_map, int* d_offsets, unsigned int* grp_cnt, int* threadToGroup, int* localIndexInGroup, int* group_map, int* group_size, int* group_load, int numBadWarps
                                , int GROUP_THRESHOLD){

    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id >= numBadWarps* 32) return;

    int node_to_be_processed = thread_map[id];


    if(threadIdx.x % 32 == 0){
        
        int curr_sz = 1;
        int curr_load = d_offsets[node_to_be_processed+1] - d_offsets[node_to_be_processed];
        int curr_groupId = atomicInc(grp_cnt, INT_MAX);
        threadToGroup[id] = curr_groupId;
        localIndexInGroup[id] = 0;

        for(int i = 1; i < 32; i++){

            int load_i = d_offsets[node_to_be_processed + 1 + i] - d_offsets[node_to_be_processed + i];

            if(curr_load > 32 || curr_load + load_i > GROUP_THRESHOLD){
                group_map[curr_groupId] = curr_groupId; // do we need this??
                group_size[curr_groupId] = curr_sz;
                group_load[curr_groupId] = curr_load;

                curr_load = 0;
                curr_groupId = atomicInc(grp_cnt, INT_MAX);
                curr_sz = 0;
            }

            curr_load += load_i;
            curr_sz++;
            threadToGroup[id + i] = curr_groupId; // using new_Id here. Correct since local mapping for threads within a warp is still the same.
            localIndexInGroup[id + i] = curr_sz - 1;

        }

        group_map[curr_groupId] = curr_groupId;
        group_size[curr_groupId] = curr_sz;
        group_load[curr_groupId] = curr_load;
    }
    
}

// do I need to sort groupMap or simply sort threadToGroupMapping based on the size...but would it be a key value pair mapping??

// calculate a prefix sum with the sizes of each group pref[grp]

// once all the groups have been sorted, id needs to be reassigned from thread_map[id] to 
// pref[group_map[threadToGroup[thread_map[id]]]] - group_size[group_map[threadToGroup[thread_map[id]]]] + localIndexInGroup[new_id]
// do I actually need to sort group_size too??

__global__ void preprocess5(int* thread_map, int numBadWarps, int* threadToGroup, int* r_map, int* pref, int* group_size, int* localIndexInGroup){
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    if( id >= numBadWarps* 32) return;

    int old_grp = threadToGroup[id];
    int new_group = r_map[old_grp];
    int grpPos = pref[new_group] - group_size[new_group]; // sizes need to be sorted to calculate prefix sum...
    
    thread_map[grpPos + localIndexInGroup[id]] = thread_map[id];

}

__global__ void make_rev_map(int* rmap, int* gmap, int N){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < N){
        rmap[gmap[id]] = id;
    }
}

// OPT KERNELS END


__global__ void initialize_graph(int N, int *parent, int *phase, int ROOT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        parent[i] = -1;
        phase[i] = -1;
        if (i == ROOT)
        {
            phase[ROOT] = 0;
        }
    }
}


__global__ void broadcast_graph(int p, int N, int *phase, int *CSR_N, int *CSR_F, int *parent, int *discovered, int* thread_mappings)
{

    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id >= N) return;

    id = thread_mappings[id];

    if (phase[id] == p)
        for (int j =  CSR_N[id]; j < CSR_N[id + 1]; j++)
        {
            int nbr = CSR_F[j];
            if (phase[nbr] < 0)
            {
                phase[nbr] = p + 1;
                parent[nbr] = id;
                *discovered = 0;
            }
        }
}

void printSolution(int *phase, int *parent, int N)
{
    for (int i = 0; i < N; i++)
    {
        printf("node %d -> parent = %d, phase = %d\n", i, parent[i], phase[i]);
    }
}   

void readGraph(char *filename, int *N, int *ROOT, int **CSR_N, int **CSR_F, int *N_sz, int *F_sz)
{ // verified

    FILE *in = fopen(filename, "r");
    fscanf(in, "%d\n", N);
    fscanf(in, "%d\n", N_sz);
    fscanf(in, "%d\n", F_sz);
    fscanf(in, "%d\n", ROOT);
    *CSR_N = (int *)malloc(*N_sz * sizeof(int));
    *CSR_F = (int *)malloc(*F_sz * sizeof(int));

    for (int i = 0; i < *N_sz; i++)
        fscanf(in, "%d ", &((*CSR_N)[i]));
    for (int i = 0; i < *F_sz; i++)
        fscanf(in, "%d ", &((*CSR_F)[i]));

    fclose(in);
}

int main(int argc, char *argv[])
{
    int N, ROOT;
    int N_sz, F_sz;
    int *CSR_F;
    int *phase, *CSR_N, p;

    readGraph(argv[1], &N, &ROOT, &CSR_N, &CSR_F, &N_sz, &F_sz);
    int *parent, *parent_d, *phase_d;
    int *CSR_N_d, *CSR_F_d;
    parent = (int *)malloc(N * sizeof(int));
    phase = (int *)malloc(N * sizeof(int));

    for (int i = 0; i < NUM_TESTS; i++)
    {


        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipMalloc((void **)&parent_d, N * sizeof(int));
        hipMalloc((void **)&phase_d, N * sizeof(int));
        hipMalloc((void **)&CSR_N_d, N_sz * sizeof(int));
        hipMalloc((void **)&CSR_F_d, F_sz * sizeof(int));

        hipMemcpy(CSR_N_d, CSR_N, N_sz * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(CSR_F_d, CSR_F, F_sz * sizeof(int), hipMemcpyHostToDevice);

        p = 0;

        int discovered = 1;
        int *discovered_d;

        hipMalloc((void **)&discovered_d, sizeof(int));
        hipMemcpy(discovered_d, &discovered, sizeof(int), hipMemcpyHostToDevice);

        int num_threads = 512;
        int num_blocks = ceil((N * 1.0) / num_threads);

        initialize_graph<<<num_blocks, num_threads>>>(N, parent_d, phase_d, ROOT);
        hipError_t err = hipGetLastError(); // Get error code

        if (err != hipSuccess)
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();

        discovered = 0;
        float milliseconds = 0;

// opt portion

    hipEventRecord(start);
    int m = N;
    int* d_row_offsets = CSR_N_d;

    int *P, *d_P, numBadWarps, *d_numBadWarps;
    bool *isBad, *d_isBad;
    int* thread_mappings;
    int *d_Good, *d_Bad, *d_Gi, *d_Bi;
    int Gi, Bi;
    int totalWarps = (m / 32);
    P = (int *)malloc(totalWarps * sizeof(int));
    isBad = (bool *)malloc(totalWarps);
    hipMalloc(&thread_mappings, m*sizeof(int));
    numBadWarps = 0;
    hipMalloc(&d_numBadWarps, sizeof(int));
    hipMemset(d_numBadWarps, 0, sizeof(int));
    hipMalloc(&d_Gi, sizeof(int));
    hipMalloc(&d_Bi, sizeof(int));
    hipMalloc(&d_P, totalWarps * sizeof(int));

    hipMalloc(&d_isBad, totalWarps);
    hipMemset(d_isBad, 0, totalWarps);

    hipMemset(d_numBadWarps, 0, sizeof(int));
    preprocess1<<<ceil(((float)totalWarps * 32) / 512), 512>>>(totalWarps * 32, d_P, d_isBad, d_row_offsets, d_numBadWarps, 350);
    CHECK(hipMemcpy(&numBadWarps, d_numBadWarps, sizeof(int), hipMemcpyDeviceToHost));
    hipMalloc(&d_Good, numBadWarps * sizeof(int));
    hipMalloc(&d_Bad, numBadWarps * sizeof(int));
    hipMemset(d_Gi, 0, sizeof(int));
    hipMemset(d_Bi, 0, sizeof(int));
    preprocess2<<<ceil(totalWarps * 1.0 / 512), 512>>>(d_isBad, numBadWarps, d_Good, d_Bad, totalWarps, d_Gi, d_Bi);
    CHECK(hipMemcpy(&Gi, d_Gi, sizeof(int), hipMemcpyDeviceToHost));
    preprocess2_1<<<max(1, (int)ceil(Gi * 1.0 / 512)), 512>>>(d_P, Gi, d_Good, d_Bad);


    preprocess3<<<num_blocks, num_threads>>>(m, numBadWarps, d_row_offsets, thread_mappings, d_P, totalWarps);

    int* hmap;
    int rmpd = numBadWarps*32;

    hmap = (int*)malloc(sizeof(int) * (rmpd + 1));
    hipMemcpy(hmap, thread_mappings, (rmpd+1)* sizeof(int), hipMemcpyDeviceToHost);

    unsigned int* grp_cnt;
    int* threadToGroup;
    int* localIndexInGroup;
    int* group_map;
    int* group_size;
    int* group_load;
    int* pref;


    hipMalloc(&grp_cnt, sizeof(int));
    hipMalloc(&threadToGroup, numBadWarps*32*sizeof(int));
    hipMalloc(&localIndexInGroup, numBadWarps*32*sizeof(int));
    hipMalloc(&group_map, numBadWarps*32*sizeof(int));
    hipMalloc(&group_size, numBadWarps*32*sizeof(int));
    hipMalloc(&group_load, numBadWarps*32*sizeof(int));


    hipMemset(grp_cnt, 0, sizeof(int));

    

    int p4block_size = 512;
    int p4blocks = (numBadWarps*32 - 1)/512 + 1;

    preprocess4<<<p4blocks, p4block_size>>>(thread_mappings, d_row_offsets, grp_cnt, threadToGroup, localIndexInGroup, group_map, group_size, group_load, numBadWarps, 50);


    int h_grp_cnt;
    hipMemcpy(&h_grp_cnt, grp_cnt, sizeof(int), hipMemcpyDeviceToHost);

    auto zipIt = thrust::make_zip_iterator(thrust::make_tuple(group_map, group_size));

    thrust::sort_by_key(thrust::device, group_load, group_load+h_grp_cnt, zipIt);

    hipMalloc(&pref, h_grp_cnt*sizeof(int));
    hipMemcpy(pref, group_size, h_grp_cnt * sizeof(int), hipMemcpyDeviceToDevice);
    thrust::inclusive_scan(thrust::device, pref, pref+h_grp_cnt, pref);

    // make reverse group mapping
    int* r_map;
    hipMalloc(&r_map, sizeof(int)* h_grp_cnt);
    make_rev_map<<<(h_grp_cnt - 1)/512 + 1, 512>>>(r_map, group_map, h_grp_cnt);

    preprocess5<<<p4blocks, p4block_size>>>(thread_mappings, numBadWarps, threadToGroup, r_map, pref, group_size, localIndexInGroup);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("opt time = %f\n", milliseconds);
    
        milliseconds = 0;
        hipEventRecord(start);

        while (!discovered)
        {

            discovered = 1;
            hipMemcpy(discovered_d, &discovered, sizeof(int), hipMemcpyHostToDevice);

            broadcast_graph<<<num_blocks, num_threads>>>(p, N, phase_d, CSR_N_d, CSR_F_d, parent_d, discovered_d, thread_mappings);

            CHECK(hipMemcpy(&discovered, discovered_d, sizeof(int), hipMemcpyDeviceToHost));
            p++;
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Runtime  = %f\n", milliseconds);

        printf("------------------------Total no. of phases : %d ------------------------\n\n\n", p);
        // printf("Avg time taken by the kernels : %f ms\n\n", avg_time / iter);
        hipMemcpy(phase, phase_d, N * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(parent, parent_d, N * sizeof(int), hipMemcpyDeviceToHost);
        // fprintf(stderr,"%d %d %d\n",phase, parent[ROOT], phase[ROOT]);
       // printSolution(phase, parent, N);
    }
    return 0;
}
