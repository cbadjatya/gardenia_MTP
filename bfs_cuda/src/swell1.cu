#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <thrust/sort.h>
#include <thrust/scan.h>


#define NUM_TESTS 1

#define CHECK(call) \
    {               \
        call;       \
    }

#define CHECK_DEBUG(call)                                                         \
    {                                                                             \
        const hipError_t error = call;                                           \
        if (error != hipSuccess)                                                 \
        {                                                                         \
            printf("Error : %s: %d -> ", __FILE__, __LINE__);                     \
            printf("code : %d, reason : %s\n", error, hipGetErrorString(error)); \
        }                                                                         \
    }

using namespace std;


void readGraph(char *filename, int *N, int *ROOT, int **row_offsets, int **column_indices, int *N_sz, int *F_sz)
{ // verified

    FILE *in = fopen(filename, "r");
    fscanf(in, "%d\n", N);
    fscanf(in, "%d\n", N_sz);
    fscanf(in, "%d\n", F_sz);
    fscanf(in, "%d\n", ROOT);
    *row_offsets = (int *)malloc(*N_sz * sizeof(int));
    *column_indices = (int *)malloc(*F_sz * sizeof(int));

    for (int i = 0; i < *N_sz; i++)
        fscanf(in, "%d ", &((*row_offsets)[i]));
    for (int i = 0; i < *F_sz; i++)
        fscanf(in, "%d ", &((*column_indices)[i]));

    fclose(in);
}

__global__ void populate(int N, int* row_offsets, int* column_indices, int* new_ind, int* tmap, int* nnz, int* warp_base, int* check){
    // __shared__ int sh_offsets[32];
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= N) return;
    // check if its a full warp

    int nid = tmap[id]; // nid is the vertex this thread is supposed to process
    int wid = id / 32;
    int wsz = min(32, N - wid*32);

    // if(N - wid*32 < 32){
    //     wsz = N - wid*32; // could be simplified
    // }

    // if(id%32 == 0){
    //     sh_offsets[wid] = 0;
    // }
    // __syncwarp();
    
    int wbase = warp_base[wid] + id%wsz; // the base address for this vertex
    // atomicAdd(&sh_offsets[wid], (int)(0 == nnz[id]));
    //  __syncwarp();
    // sh_offsets[wid] += 0 == nnz[id];
    // for (int iw = 16; iw > 0; iw = iw / 2)
    // {
    //     sh_offset = max(sh_offset, __shfl_down_sync(-1, sh_offset, iw));
    // }
    // __syncwarp();

    // for(int i = 0;i<nnz[id];){
    //     int ind = column_indices[i + row_offsets[nid]];
    //     new_ind[wbase] = ind;
    //     // atomicAdd(&d_check[wbase],1);
    //     wbase -= sh_offsets[wid];
    //     // warp shuffle, find maximum sh_offset in the warp
    //     i++;
    //     __syncwarp();
    //     atomicAdd(&sh_offsets[wid], (int)(i == nnz[id]));
    //     // sh_offsets[wid] += i == nnz[id];
    //     __syncwarp();
    //     wbase += wsz;
    // }
    int row_begin = row_offsets[nid];
    int row_end = row_offsets[nid + 1];

    for(int i = row_begin; i<row_end; i++){
        int ind = column_indices[i];
        new_ind[wbase] = ind;
        atomicAdd(&check[wbase],1);
        // wbase -= sh_offsets[wid];
        // warp shuffle, find maximum sh_offset in the warp
        // i++;
        // __syncwarp();
        // atomicAdd(&sh_offsets[wid], (int)(i == nnz[id]));
        // sh_offsets[wid] += i == nnz[id];
        // __syncwarp();
        wbase += wsz;
    }

    // this mapping might be called ELL-32-C (or some such thing where 32 is sigma)

}

__global__ void initialize_graph(int N, int *parent, int *phase, int ROOT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        parent[i] = -1;
        phase[i] = -1;
        if (i == ROOT)
        {
            phase[ROOT] = 0;
        }
    }
}

// nnz has neighbors stored in the sorted manner. access using id not nid.
__global__ void broadcast_graph_swell(int p, int N, int *phase, int * new_column_indices, int *parent, int *discovered, int* tmap, int* warp_base, int* nnz)
{

    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id >= N) return;

    int nid = tmap[id];

    if (phase[nid] != p) return;

    int wid = id/32;
    int wsz = min(32, N - wid*32);
    int wbase = warp_base[wid] + id%wsz;

    int num_neighbors = nnz[id];

    
    for (int j =  0; j < num_neighbors; j++)
    {
        int nbr = new_column_indices[wbase];
        if (phase[nbr] < 0)
        {
            phase[nbr] = p + 1;
            parent[nbr] = nid;
            *discovered = 0;
        }
        wbase += wsz;
    }
}


__global__ void broadcast_graph_base(int p, int N, int *phase, int *row_offsets, int *column_indices, int *parent, int *discovered)
{

    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id >= N) return;

    if (phase[id] != p) return;

    int start = row_offsets[id];
    int end = row_offsets[id+1];

    for (int j =  start; j < end; j++)
    {
        int nbr = column_indices[j];
        if (phase[nbr] < 0)
        {
            phase[nbr] = p + 1;
            parent[nbr] = id;
            *discovered = 0;
        }
    }
}

int main(int argc, char *argv[])
{
    int N, ROOT;
    int N_sz, F_sz;
    int *column_indices;
    int *phase_base, *row_offsets, p, *phase_swell;

    readGraph(argv[1], &N, &ROOT, &row_offsets, &column_indices, &N_sz, &F_sz);
    
    int *parent, *parent_d, *phase_d;
    int *row_offsets_d, *column_indices_d;
    parent = (int *)malloc(N * sizeof(int));
    phase_swell = (int *)malloc(N * sizeof(int));
    phase_base = (int *)malloc(N * sizeof(int));

    // sort nodes in increasing order of number of neighbors

    int* nnz;
    int* tmap;

    nnz = (int*)malloc(N * sizeof(int)); // number of non zeroes (no. of neighbors for each node)
    
    tmap = (int*)malloc(N * sizeof(int)); // remapping id for each vertex

    for(int i=0;i<N;i++){
        nnz[i] = row_offsets[i+1] - row_offsets[i];
        tmap[i] = i;
    }

    thrust::sort_by_key(tmap, tmap + N, nnz, thrust::greater<int>()); // nnz and tmap are both sorted

    int* warp_sizes;
    int num_warps = (N-1)/32 + 1;
    warp_sizes = (int*)malloc(sizeof(int) * num_warps);
    int tot_size = 0;
    for(int i=0;i<N;i+=32){
        // int sz = 0;
        // for(int j = 0; j < 32; j++){
        //     sz += nnz[i+j];
        // }
        warp_sizes[i/32] = nnz[i]*32;
        tot_size += nnz[i]*32;
    }
    // the last warp won't be fully filled
    if(N%32 != 0){
        int x = (N/32) * 32;
        // int sz = 0;
        // for(;x < N; x++){
        //     sz += nnz[x];
        // }
         warp_sizes[N/32 + 1] = nnz[x]*(N-x);
         tot_size += nnz[x]*(N-x);
    }
    
    int* warp_base = (int*)malloc(sizeof(int) * num_warps);
    thrust::exclusive_scan(warp_sizes, warp_sizes + num_warps, warp_base);

    int* d_row_offsets, *d_column_indices, *d_new_ind, *d_tmap, *d_nnz, *d_warp_base;
    hipMalloc(&d_row_offsets, (N+1) * sizeof(int));
    hipMalloc(&d_column_indices, F_sz * sizeof(int));
    hipMalloc(&d_new_ind, tot_size * sizeof(int));
    hipMalloc(&d_tmap, N * sizeof(int));
    hipMalloc(&d_nnz, N * sizeof(int));
    hipMalloc(&d_warp_base, num_warps * sizeof(int));

    hipMemcpy(d_row_offsets, row_offsets, (N+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_column_indices, column_indices, F_sz * sizeof(int), hipMemcpyHostToDevice);
    // hipMemcpy(d_new_ind, new_ind, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_tmap, tmap, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_nnz, nnz, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_warp_base, warp_base, num_warps * sizeof(int), hipMemcpyHostToDevice);

    int num_blocks = (N-1)/512 + 1;
    int num_threads = 512;

    // implementing a memory check...was any index in d_new_ind accessed more/less than once
    int* check = (int*)malloc(sizeof(int) * tot_size);
    int* d_check;
    hipMalloc(&d_check, sizeof(int) * tot_size);
    hipMemset(d_check, 0, sizeof(int) * tot_size);


    populate<<<num_blocks,512>>>(N, d_row_offsets, d_column_indices, d_new_ind, d_tmap, d_nnz, d_warp_base, d_check); // add d_check if necessary
    hipMemcpy(check,d_check, tot_size * sizeof(int), hipMemcpyDeviceToHost);
    bool check_failed = false;
    for(int i=0; i<tot_size; i++){
        if(check[i] > 1){
            check_failed = true;
            cout<<i<<" "<<check[i]<<"\n";
        }
    }
    if(check_failed == true) return 0;

        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipMalloc((void **)&parent_d, N * sizeof(int));
        hipMalloc((void **)&phase_d, N * sizeof(int));
        hipMalloc((void **)&row_offsets_d, N_sz * sizeof(int));
        hipMalloc((void **)&column_indices_d, F_sz * sizeof(int));

        hipMemcpy(row_offsets_d, row_offsets, N_sz * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(column_indices_d, column_indices, F_sz * sizeof(int), hipMemcpyHostToDevice);

        p = 0;

        int discovered = 1;
        int *discovered_d;

        hipMalloc((void **)&discovered_d, sizeof(int));
        hipMemcpy(discovered_d, &discovered, sizeof(int), hipMemcpyHostToDevice);

        // int num_threads = 512;
        // int num_blocks = ceil((N * 1.0) / num_threads);

        initialize_graph<<<num_blocks, num_threads>>>(N, parent_d, phase_d, ROOT);
        hipError_t err = hipGetLastError(); // Get error code

        if (err != hipSuccess)
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();

        discovered = 0;
        float milliseconds = 0;

        hipEventRecord(start);

        while (!discovered)
        {

            discovered = 1;
            hipMemcpy(discovered_d, &discovered, sizeof(int), hipMemcpyHostToDevice);

            broadcast_graph_base<<<num_blocks, num_threads>>>(p, N, phase_d, row_offsets_d, column_indices_d, parent_d, discovered_d);

            CHECK(hipMemcpy(&discovered, discovered_d, sizeof(int), hipMemcpyDeviceToHost));
            p++;
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Base Runtime = %f\n", milliseconds);
        printf("------------------------Total no. of phases : %d ------------------------\n\n\n", p);
        // printf("Avg time taken by the kernels : %f ms\n\n", avg_time / iter);
        hipMemcpy(phase_base, phase_d, N * sizeof(int), hipMemcpyDeviceToHost);


        initialize_graph<<<num_blocks, num_threads>>>(N, parent_d, phase_d, ROOT);
        err = hipGetLastError(); // Get error code

        if (err != hipSuccess)
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
            exit(-1);
        }
        hipDeviceSynchronize();

        discovered = 0;
        milliseconds = 0;

        p = 0;

        hipEventRecord(start);

        while (!discovered)
        {

            discovered = 1;
            hipMemcpy(discovered_d, &discovered, sizeof(int), hipMemcpyHostToDevice);

            broadcast_graph_swell<<<num_blocks, num_threads>>>(p, N, phase_d, d_new_ind, parent_d, discovered_d, d_tmap, d_warp_base, d_nnz);

            CHECK(hipMemcpy(&discovered, discovered_d, sizeof(int), hipMemcpyDeviceToHost));
            p++;
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Swell Runtime = %f\n", milliseconds);
        printf("------------------------Total no. of phases : %d ------------------------\n\n\n", p);
        // printf("Avg time taken by the kernels : %f ms\n\n", avg_time / iter);
        hipMemcpy(phase_swell, phase_d, N * sizeof(int), hipMemcpyDeviceToHost);


        // comparison

        for(int i=0;i<N;i++){
            if(phase_swell[i] != phase_base[i]){
                cout<<i<<" "<<phase_swell[i]<<" "<<phase_base[i]<<"\n";
                cout<<"Incorrect\n";
                return 0;
            }
        }
        cout<<"fuck yeah\n";
        return 0;














        // hipMemcpy(parent, parent_d, N * sizeof(int), hipMemcpyDeviceToHost);
        // fprintf(stderr,"%d %d %d\n",phase, parent[ROOT], phase[ROOT]);
       // printSolution(phase, parent, N);

    // hipMemcpy(check, d_check, F_sz * sizeof(int), hipMemcpyDeviceToHost);

    // for(int i=0;i<F_sz;i++){
    //     if(check[i]!=1)
    //         cout<<i<<" "<<check[i]<<"\n";
    // }

   

    // int* h_new_ind = (int*)malloc(sizeof(int) * F_sz); // F_sz is the real nnz in CSR format

    // hipMemcpy(h_new_ind, d_new_ind, F_sz * sizeof(int), hipMemcpyDeviceToHost);





    return 0;
}
